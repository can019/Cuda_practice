#include "hip/hip_runtime.h"


#include <stdio.h>
__global__ void threadCounting_atomicGlobal(int *a){
	atomicAdd(a, 1);
}

int main(void){
	int a = 0;
	int *d;

	hipMalloc((void**)&d, sizeof(int));
	hipMemset(d, 0, sizeof(int)*1);

	threadCounting_atomicGlobal<<<10240,512>>>(d);
	hipDeviceSynchronize();
	
	hipMemcpy(&a, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n",a);
	hipFree(d);
}
