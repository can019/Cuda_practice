#include "hip/hip_runtime.h"


#include <stdio.h>
__global__ void threadCounting_noSync(int *a){
	(*a)++;
}

int main(void){
	int a = 0;
	int *d;

	hipMalloc((void**)&d, sizeof(int));
	hipMemset(d, 0, sizeof(int)*1);

	threadCounting_noSync<<<10240,512>>>(d);
	hipDeviceSynchronize();
	
	hipMemcpy(&a, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n",a);
	hipFree(d);
}
