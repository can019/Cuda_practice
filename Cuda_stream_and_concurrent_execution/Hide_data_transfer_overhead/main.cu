#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define NUM_BLOCK (128*1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B*NUM_BLOCK)
#define NUM_STREAMS 2

int main(void)
{
	int *in = NULL, *out = NULL, *dIn = NULL, *dOut = NULL;

	hipHostMalloc(&in, sizeof(int)*ARRAY_SIZE); memset(in, 0, sizeof(int)*ARRAY_SIZE);
	hipHostMalloc(&out, sizeof(int)*ARRAY_SIZE); memset(out, 0, sizeof(int)*ARRAY_SIZE);
	
	hipMalloc(&dIn, sizeof(int)*ARRAY_SIZE);
	hipMalloc(&dOut, sizeof(int)*ARRAY_SIZE);
	
	LOOP_I(ARRAY_SIZE);
	in[i] = rand() % 10;
	
	// Single stream version
	hipMemcpy(dIn, in, sizeof(int)*ARRAY_SIZE, hipMemcpyHostToDevice);
	myKernel << <NUM_BLOCK, NUM_T_IN_B>>> (dIn, dOut);
	hipMemcpy(out, dOut, sizeof(int)*ARRAY_SIZE, hipMemcpyDeviceToHost);

	// Multi-stream version
	hipStream_t stream[NUM_STREAMS];
	LOOP_I(NUM_STREAMS);
	hipStreamCreate(&stream[i]);
	
	int chunkSize = ARRAY_SIZE / NUM_STREAMS;
	LOOP_I(NUM_STREAMS)
	{
		int offset = chunkSize * i;
		hipMemcpyAsync(dIn + offset, in + offset
				, sizeof(int)*chunkSize, hipMemcpyHostToDevice, stream[i]);
		myKernel <<<NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i]>>>
			(dIn + offset, dOut + offset);
		hipMemcpyAsync(out2 + offset, dOut + offset
				, sizeof(int)*chunkSize, hipMemcpyDeviceToHost, stream[i]);
	}
	hipDeviceSynchronize();
	
	LOOP_I(NUM_STREAMS) hipStreamDestroy(stream[i]);
	hipFree(dIn); hipFree(dOut);
	hipHostFree(in); hipHostFree(out); hipHostFree(out2);
}
