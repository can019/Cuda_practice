#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//#define NUM_DATA = 65535
unsigned NUM_DATA = 2147483647;
//#define MAX_THREAD_IN_SINGLE_BLOCK = 8*8*8
//#define MAX_BLOCK_COUNT_IN_GRID = 1024*1024*1024*4 // Same as unsigned size
//#define BLOCK_SIZE = 1024
//#define NUM_THREAD_IN_BLOCK = 8*8*8
//int block_size = 1024;
__global__ void vecAdd(unsigned *_a, unsigned *_b, unsigned *_c)
{
	//int tID = threadId.x; //1차원 grid 1차원 block
	//int tID = threadIdx.y*blockDim.x+threadIdx.x; //1차원 grid 2차원 block
	//int tID = (blockDim.x*blockDim.y*threadIdx.z) + //1차원 grid 3차원 block
	//	(threadIdx.y*blockDim.x)+
	//		threadIdx.x
	//중략

	unsigned tID = blockIdx.z*(gridDim.y*gridDim.x*1)
		+blockIdx.y*(gridDim.x*1)+blockIdx.x*(blockDim.x*blockDim.y*blockDim.z)+blockDim.y*blockDim.x*threadIdx.z+blockDim.x*threadIdx.y+threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void){
	unsigned *a, *b, *c;
	unsigned *d_a, *d_b, *d_c;
	
	unsigned memSize = NUM_DATA*sizeof(unsigned);
	printf("%u elements, memSize = %u bytes\n", NUM_DATA, memSize);


	a = new unsigned[NUM_DATA]; memset(a, 0, memSize);
	b = new unsigned[NUM_DATA]; memset(b, 0, memSize);
	c = new unsigned[NUM_DATA]; memset(c, 0, memSize);

	for (unsigned i =0; i< NUM_DATA; i++){
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}
	hipMalloc(&d_a, memSize);
	hipMalloc(&d_b, memSize);
	hipMalloc(&d_c, memSize);

	//Under two line synchronize automatically. You don't need to use synchronize.
	hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);

	// Kernel call
	dim3 dimGrid(2147483647, 1, 1);
	dim3 dimBlock(1, 1, 1); //dimBlock should be total <= 1024
	vecAdd<<<dimGrid, dimBlock >>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);

	//check results
	bool result = true;
	for (unsigned i =0; i<NUM_DATA; i++)
	{
		if((a[i] + b[i]) != c[i]){
			printf("[%u] The results is not matched! (%u, %u)\n",
					i, a[i] + b[i], c[i]);
			result = false;
		}
	}

	if(result)
		printf("GPU works well!\n");

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	delete [] a; delete [] b; delete [] c;


	return 0;
}
