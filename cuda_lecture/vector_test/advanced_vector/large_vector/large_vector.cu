#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_DATA 1024000 // << 1024*100
#define MAX_THREAD_IN_SINGLE_BLOCK = 1024

__global__ void vecAdd(int *_a, int *_b, int *_c)
{
	int tID = blockIdx.x*blockDim.x+threadIdx.x;
	_c[tID] = _a[tID] + _b[tID];
}

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int memSize = sizeof(int)*NUM_DATA;
	printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

	a = new int[NUM_DATA]; memset(a, 0, memSize);
	b = new int[NUM_DATA]; memset(b, 0, memSize);
	c = new int[NUM_DATA]; memset(c, 0, memSize);
	
	for (int i =0; i< NUM_DATA; i++){
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}
	hipMalloc(&d_a, memSize);
	hipMalloc(&d_b, memSize);
	hipMalloc(&d_c, memSize);
	
	//Under two line synchronize automatically. You don't need to use synchronize.
	hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice); 
	hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);
	
	// Kernel call
	dim3 dimGrid(NUM_DATA/1024, 1, 1);
	dim3 dimBlock(1024,1,1); //MAX_SIZE = 1024
	vecAdd<<<dimGrid, dimBlock >>>(d_a, d_b, d_c);
	hipDeviceSynchronize();
	hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);

	//check results
	bool result = true;
	for (int i =0; i<NUM_DATA; i++)
	{
		if((a[i] + b[i]) != c[i]){
			printf("[%d] The results is not matched! (%d, %d)\n",
					i, a[i] + b[i], c[i]);
			result = false;
		}
	}

	if(result)
		printf("GPU works well!\n");

	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	delete [] a; delete [] b; delete [] c;

	return 0;
}
