#include "hip/hip_runtime.h"


#include <stdio.h>
__global__ void threadCounting_atomicShared(int *a){
	__shared__ int sa;
	if(threadIdx.x == 0)
		sa = 0;
	__syncthreads();

	atomicAdd(&sa, 1);
	__syncthreads();

	if(threadIdx.x ==0)
		atomicAdd(a, sa);
}

int main(void){
	int a = 0;
	int *d;

	hipMalloc((void**)&d, sizeof(int));
	hipMemset(d, 0, sizeof(int)*1);

	threadCounting_atomicShared<<<10240,512>>>(d);
	hipDeviceSynchronize();

	hipMemcpy(&a, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d\n",a);
	hipFree(d);
}
