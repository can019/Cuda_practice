#include "hip/hip_runtime.h"
#include "device_launch_parmeters.h"

#include <DS_timer.h>

#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_CPU_THREADS(4)

#define ROW_SIZE(32)
#define K_SIZE(128)
#define COL_SIZE(32)

#define WORK_LOAD(1024)
#define MAT_SIZE_A (ROW_SIZE*K_SIZE)
#define MAT_SIZE_B (K_SIZE*COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE*COL_SIZE)

float A[ROW_SIZE][K_SIZE];
float B[K_SIZE][COL_SIZE];

// Timer
#define TIMER_HOST	0
#define TIMER_KERNEL	1

